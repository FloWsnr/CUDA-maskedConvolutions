#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;


__global__ void dot_product(float* out, float* v1, float* v2, int n) {

    cg::thread_block cta = cg::this_thread_block();

    int t_idx = threadIdx.x; // thread index
    int b_dim = blockDim.x; // number of threads per block
    int b_idx = blockIdx.x; // block index

    int idx = b_idx * b_dim + t_idx;
    int stride = b_dim * gridDim.x; // total number of threads

    // make shared memory the next power of 2 multiple of block size
    extern __shared__ float sdata[]; // shared memory for intermediate results


    float sum = 0.0f;
    for (int i = idx; i < n; i += stride) {
        // each thread computes one element (or multiple)
        // of the output and already computes the sum of these elements
        sum += v1[i] * v2[i];
    }
    sdata[t_idx] = sum; // partial sum is stored in shared memory
    cg::sync(cta); // wait for all threads to finish

    // reduction
    for (int stride = b_dim / 2; stride > 0; stride /= 2) {
        // the lower half of threads per block perform the reduction
        // afterwards, the stride is divided by two and repeat
        // until all values are summed in element 0 of the shared memory

        // Example:
        // block dim: 256
        // -> stride: 128
        // -> sdata[0] = sdata[0] + sdata[128]
        // -> sdata[1] = sdata[1] + sdata[129]

        // Next iteration:
        // -> stride: 64
        // -> sdata[0] = sdata[0] + sdata[64]
        // -> sdata[1] = sdata[1] + sdata[65]

        if (t_idx < stride) {
            sdata[t_idx] += sdata[t_idx + stride];
        }
        cg::sync(cta); // wait for all threads to finish
    }

    if (t_idx == 0) {
        atomicAdd(out, sdata[0]);
    }
}

__global__ void convolution_1d(float* out, float* arr, float* kernel, int n_arr, int n_kernel, bool* mask, float pad_val) {

    int t_idx = threadIdx.x; // thread index
    int b_dim = blockDim.x; // number of threads per block
    int b_idx = blockIdx.x; // block index

    int idx = b_idx * b_dim + t_idx;
    int stride = b_dim * gridDim.x; // total number of threads

    for (int i = idx; i < n_arr; i += stride) {
        out[i] = 0.0f;

        // loop over kernel
        for (int j = 0; j < n_kernel; ++j) {
            if (i - j >= 0 && i - j < n_arr) {
                out[i] += arr[i - j] * kernel[j];
            }
            else {
                out[i] += pad_val * kernel[j];
            }
        }
    }


}

int main() {
    int vector_size = 10000000;
    int kernel_size = 100;


    // TODO: use deviceQuery to get best block size
    int block_size = 256;
    // rounded up to nearest multiple of block size
    int num_blocks = (vector_size + block_size - 1) / block_size;

    // Declare variables
    float* v1;
    float* kernel;
    float* v_out;
    bool* mask;

    // Allocate memory
    hipMallocManaged(&v1, vector_size * sizeof(float));
    hipMallocManaged(&kernel, vector_size * sizeof(float));
    hipMallocManaged(&v_out, vector_size * sizeof(float));
    hipMallocManaged(&mask, kernel_size * sizeof(bool));

    // Initialize variables
    for (int i = 0; i < vector_size; i++) {
        v1[i] = 1.0f;
    }

    for (int i = 0; i < kernel_size; i++) {
        kernel[i] = 1.0f;
        if (i % 2 == 0) {
            mask[i] = true;
        }
        else {
            mask[i] = false;
        }
    }

    // Launch kernel
    convolution_1d << < num_blocks, block_size, block_size * sizeof(float) >> > (v_out, v1, kernel, vector_size, kernel_size, mask, 0.0f);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Print result
    // should be 2 * N
    std::cout << v_out[50] << std::endl;


    // Free memory
    hipFree(v1);
    hipFree(kernel);
    hipFree(v_out);
    hipFree(mask);

    return 0;
}
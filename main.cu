#include "hip/hip_runtime.h"
/*******************************************************************************
* Kernel to compute the masked 3D convolution of a tensor and a kernel.
* The convolution is done with padding and the output is 'same', i.e. the
* output is the same size as the input.
*
*/

#include <iostream>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;


__global__ void dot_product(float* out, float* v1, float* v2, int n) {

    cg::thread_block cta = cg::this_thread_block();

    int t_idx = threadIdx.x; // thread index
    int b_dim = blockDim.x; // number of threads per block
    int b_idx = blockIdx.x; // block index

    int idx = b_idx * b_dim + t_idx;
    int stride = b_dim * gridDim.x; // total number of threads

    // make shared memory the next power of 2 multiple of block size
    extern __shared__ float sdata[]; // shared memory for intermediate results


    float sum = 0.0f;
    for (int i = idx; i < n; i += stride) {
        // each thread computes one element (or multiple)
        // of the output and already computes the sum of these elements
        sum += v1[i] * v2[i];
    }
    sdata[t_idx] = sum; // partial sum is stored in shared memory
    cg::sync(cta); // wait for all threads to finish

    // reduction
    for (int stride = b_dim / 2; stride > 0; stride /= 2) {
        // the lower half of threads per block perform the reduction
        // afterwards, the stride is divided by two and repeat
        // until all values are summed in element 0 of the shared memory

        // Example:
        // block dim: 256
        // -> stride: 128
        // -> sdata[0] = sdata[0] + sdata[128]
        // -> sdata[1] = sdata[1] + sdata[129]

        // Next iteration:
        // -> stride: 64
        // -> sdata[0] = sdata[0] + sdata[64]
        // -> sdata[1] = sdata[1] + sdata[65]

        if (t_idx < stride) {
            sdata[t_idx] += sdata[t_idx + stride];
        }
        cg::sync(cta); // wait for all threads to finish
    }

    if (t_idx == 0) {
        atomicAdd(out, sdata[0]);
    }
}

__global__ void convolution_1d(
    float* out,
    float* arr,
    int n_arr,
    float* kernel,
    int n_kernel,
    bool* mask,
    float pad_val) {

    int t_idx = threadIdx.x; // thread index
    int b_dim = blockDim.x; // number of threads per block
    int b_idx = blockIdx.x; // block index

    int idx = b_idx * b_dim + t_idx;
    int stride = b_dim * gridDim.x; // total number of threads

    for (int i = idx; i < n_arr; i += stride) {
        out[i] = 0.0f;

        // loop over kernel
        for (int j = 0; j < n_kernel; ++j) {

            // index the array with implicitreversed kernel
            int input_index = i - j + n_kernel / 2;

            if (input_index >= 0 && input_index < n_arr) {
                out[i] += arr[input_index] * kernel[j];
            }
            else {
                out[i] += pad_val * kernel[j];
            }
        }
    }
}

int main() {
    int vector_size = 3;
    int kernel_size = 3;

    // TODO: use deviceQuery to get best block size
    int block_size = 256;
    // rounded up to nearest multiple of block size
    int num_blocks = (vector_size + block_size - 1) / block_size;

    // Declare variables
    float* v1;
    float* kernel;
    float* v_out;
    bool* mask;

    // Allocate memory
    hipMallocManaged(&v1, vector_size * sizeof(float));
    hipMallocManaged(&mask, vector_size * sizeof(bool));
    hipMallocManaged(&kernel, kernel_size * sizeof(float));
    hipMallocManaged(&v_out, vector_size * sizeof(float));

    for (int i = 0; i < vector_size; i++) {
        v1[i] = i + 1.0f;
        v_out[i] = 0.0f;
    }
    for (int i = 0; i < kernel_size; i++) {
        kernel[i] = 4.0f + i;
    }


    // Initialize variables
    // for (int i = 0; i < vector_size; i++) {
    //     v1[i] = 4.0f;
    //     if (i % 2 == 0) {
    //         mask[i] = true;
    //     }
    //     else {
    //         mask[i] = false;
    //     }
    // }

    // for (int i = 0; i < kernel_size; i++) {
    //     kernel[i] = 1.0f / kernel_size;
    // }

    // Launch kernel
    convolution_1d << < num_blocks, block_size >> > (v_out, v1, vector_size, kernel, kernel_size, mask, 0.0f);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Print result
    for (int i = 0; i < vector_size; i++) {
        std::cout << v_out[i] << " ";
    }
    std::cout << std::endl;


    // Free memory
    hipFree(v1);
    hipFree(kernel);
    hipFree(v_out);
    hipFree(mask);

    return 0;
}
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <gtest/gtest.h>
#include "masked_conv.hpp"

TEST(Conv1D, TestNormalConvWorks) {
  int vector_size = 3;
  int kernel_size = 3;

  // TODO: use deviceQuery to get best block size
  int block_size = 3;
  // rounded up to nearest multiple of block size
  int num_blocks = 1;

  // Declare variables
  float* v1;
  float* kernel;
  float* v_out;
  bool* mask;

  // Allocate memory
  hipMallocManaged(&v1, vector_size * sizeof(float));
  hipMallocManaged(&mask, vector_size * sizeof(bool));
  hipMallocManaged(&kernel, kernel_size * sizeof(float));
  hipMallocManaged(&v_out, vector_size * sizeof(float));

  for (int i = 0; i < vector_size; i++) {
    v1[i] = i + 1.0f;
    v_out[i] = 0.0f;
    mask[i] = true;
  }
  for (int i = 0; i < kernel_size; i++) {
    kernel[i] = 4.0f + i;
  }

  // Launch kernel
  convolution_1d(num_blocks, block_size, v_out, v1, vector_size, kernel, kernel_size, mask, 0.0f);

  // Wait for kernel to finish
  hipDeviceSynchronize();

  // Check results
  EXPECT_EQ(v_out[0], 13.0f);
  EXPECT_EQ(v_out[1], 28.0f);
  EXPECT_EQ(v_out[2], 27.0f);

  // Free memory
  hipFree(v1);
  hipFree(kernel);
  hipFree(v_out);
  hipFree(mask);
}

TEST(Conv1D, TestMaskedConvWorks) {
  int vector_size = 3;
  int kernel_size = 3;

  // TODO: use deviceQuery to get best block size
  int block_size = 3;
  // rounded up to nearest multiple of block size
  int num_blocks = 1;

  // Declare variables
  float* v1;
  float* kernel;
  float* v_out;
  bool* mask;

  // Allocate memory
  hipMallocManaged(&v1, vector_size * sizeof(float));
  hipMallocManaged(&mask, vector_size * sizeof(bool));
  hipMallocManaged(&kernel, kernel_size * sizeof(float));
  hipMallocManaged(&v_out, vector_size * sizeof(float));

  for (int i = 0; i < vector_size; i++) {
    v1[i] = i + 1.0f;
    v_out[i] = 0.0f;
    mask[i] = true;
  }
  for (int i = 0; i < kernel_size; i++) {
    kernel[i] = 4.0f + i;
  }

  // Set first element of mask to false
  mask[0] = false;

  // Launch kernel
  convolution_1d(num_blocks, block_size, v_out, v1, vector_size, kernel, kernel_size, mask, 0.0f);

  // Wait for kernel to finish
  hipDeviceSynchronize();

  // Check results
  EXPECT_EQ(v_out[0], 0.0f); // first element is skipped
  EXPECT_EQ(v_out[1], 22.0f);
  EXPECT_EQ(v_out[2], 27.0f);

  // Free memory
  hipFree(v1);
  hipFree(kernel);
  hipFree(v_out);
  hipFree(mask);
}

TEST(Conv3D, TestNormalConvWorks) {

  /* Test normal 3D convolution with a uniform kernel
  * Input and kernel are both filled with 1s as element
  * Thus, the middle element of the output should be 27
  */

  int nx_arr = 5;
  int ny_arr = 5;
  int nz_arr = 5;
  int nx_kernel = 3;
  int ny_kernel = 3;
  int nz_kernel = 3;
  float pad_val = 0.0f;

  // Total number of elements
  int n_arr = nx_arr * ny_arr * nz_arr;
  int n_mask = nx_arr * ny_arr * nz_arr;
  int n_kernel = nx_kernel * ny_kernel * nz_kernel;
  int n_out = nx_arr * ny_arr * nz_arr;

  int block_size = n_arr;
  int grid_size = 1;

  // Declare variables
  float* v1;
  float* kernel;
  float* v_out;
  bool* mask;

  // Allocate memory
  hipMallocManaged(&v1, n_arr * sizeof(float));
  hipMallocManaged(&mask, n_mask * sizeof(bool));
  hipMallocManaged(&kernel, n_kernel * sizeof(float));
  hipMallocManaged(&v_out, n_out * sizeof(float));

  for (int i = 0; i < n_arr; i++) {
    v1[i] = 1.0f;
    v_out[i] = 0.0f;
    mask[i] = true;
  }
  for (int i = 0; i < n_kernel; i++) {
    kernel[i] = 1.0f;
  }

  // Launch kernel
  convolution_3d(grid_size, block_size, v_out, v1, nx_arr, ny_arr, nz_arr, kernel, nx_kernel, ny_kernel, nz_kernel, mask, pad_val);

  // Wait for kernel to finish
  hipDeviceSynchronize();

  // Check results
  EXPECT_EQ(v_out[13], 27.0f);
  EXPECT_EQ(v_out[0], 7.0f);

  // Free memory
  hipFree(v1);
  hipFree(kernel);
  hipFree(v_out);
  hipFree(mask);

}

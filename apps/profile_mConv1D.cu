#include <chrono>
#include <iostream>
#include <functional>
#include "mConv.hpp"

auto benchmark_cpu(int vector_size, int kernel_size, int num_trials) {

    // Declare variables
    float* v1{ new float[vector_size] };
    float* kernel{ new float[kernel_size] };
    float* v_out{ new float[vector_size] };
    bool* mask{ new bool[vector_size] };

    for (int i = 0; i < vector_size; i++) {
        v1[i] = i + 1.0f;
        v_out[i] = 0.0f;

        // Set every other element of mask to true
        if (i % 2 == 0) {
            mask[i] = true;
        }
        else {
            mask[i] = false;
        }
    }
    for (int i = 0; i < kernel_size; i++) {
        kernel[i] = 4.0f + i;
    }

    auto total_duration = std::chrono::nanoseconds::zero();
    // Launch
    for (int i = 0; i < num_trials; i++) {

        auto start = std::chrono::high_resolution_clock::now();
        convolution_1d_cpu(v_out, v1, vector_size, kernel, kernel_size, mask, 0.0f);
        // Record time
        auto end = std::chrono::high_resolution_clock::now();
        total_duration += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    }

    auto duration = total_duration.count() / num_trials;

    delete[] v1;
    delete[] kernel;
    delete[] v_out;
    delete[] mask;

    return duration;
}

auto benchmark_gpu(int vector_size, int kernel_size, int num_trials) {
    float* v1;
    float* kernel;
    float* v_out;
    bool* mask;

    // Allocate memory
    hipMallocManaged(&v1, vector_size * sizeof(float));
    hipMallocManaged(&mask, vector_size * sizeof(bool));
    hipMallocManaged(&kernel, kernel_size * sizeof(float));
    hipMallocManaged(&v_out, vector_size * sizeof(float));

    for (int i = 0; i < vector_size; i++) {
        v1[i] = i + 1.0f;
        v_out[i] = 0.0f;

        // Set every other element of mask to true
        if (i % 2 == 0) {
            mask[i] = true;
        }
        else {
            mask[i] = false;
        }
    }
    for (int i = 0; i < kernel_size; i++) {
        kernel[i] = 4.0f + i;
    }

    auto total_duration = std::chrono::nanoseconds::zero();
    // Launch
    for (int i = 0; i < num_trials; i++) {

        auto start = std::chrono::high_resolution_clock::now();
        // Launch kernel
        convolution_1d(v_out, v1, vector_size, kernel, kernel_size, mask, 0.0f);
        // Wait for kernel to finish
        hipDeviceSynchronize();
        // Record time
        auto end = std::chrono::high_resolution_clock::now();
        total_duration += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    }
    auto duration = total_duration.count() / num_trials;

    hipFree(v1);
    hipFree(kernel);
    hipFree(v_out);
    hipFree(mask);

    return duration;
}


int main() {


    const int num_trials = 100;
    int vector_size = 100024;
    int kernel_size = 100;

    // Max concurrent threads = SM Count * threads per SM
    // = 68 * 1536 (for RTX 3080)

    auto cpu_duration = benchmark_cpu(vector_size, kernel_size, num_trials);
    std::cout << "CPU: Average time measured: " << cpu_duration << " nanoseconds." << std::endl;

    auto gpu_duration = benchmark_gpu(vector_size, kernel_size, num_trials);
    std::cout << "GPU: Average time measured: " << gpu_duration << " nanoseconds." << std::endl;
}
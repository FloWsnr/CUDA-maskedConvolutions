#include "hip/hip_runtime.h"
/*******************************************************************************
* Kernel to compute the masked 3D convolution of a tensor and a kernel.
* The convolution is done with padding and the output is 'same', i.e. the
* output is the same size as the input.
*
*/
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include "mConv.hpp"

namespace cg = cooperative_groups;

__device__ void load_to_shared(
    float* sdata,
    const float* arr,
    const int n_arr,
    const int k_radius) {

    const unsigned int t_idx{ threadIdx.x }; // thread index
    const unsigned int b_dim{ blockDim.x }; // number of threads per block
    const unsigned int b_idx{ blockIdx.x }; // block index

    const unsigned int idx{ b_idx * b_dim + t_idx };

    for (int i = idx; i < k_radius + blockDim.x; i += blockDim.x) {
        sdata[i] = arr[i - k_radius];
    }
    __syncthreads();
}

__global__ void convolution_1d_kernel(
    float* out,
    const float* arr,
    const int n_arr,
    const float* kernel,
    const int n_kernel,
    const bool* mask,
    const float pad_val) {


    const unsigned int t_idx{ threadIdx.x }; // thread index
    const unsigned int b_dim{ blockDim.x }; // number of threads per block
    const unsigned int b_idx{ blockIdx.x }; // block index

    const unsigned int idx{ b_idx * b_dim + t_idx };
    const unsigned int stride{ b_dim * gridDim.x }; // total number of threads

    const unsigned int k_radius = n_kernel / 2;

    // shared memory for the kernel (size = k_radius + b_dim)
    extern __shared__ float sdata[];
    load_to_shared(sdata, arr, n_arr, k_radius);

    for (auto i = idx; i < n_arr; i += stride) {
        if (!mask[i]) continue; // skip if mask is false

        // loop over kernel
        for (int j = 0; j < n_kernel; ++j) {

            // index the array with implicit reversed kernel
            int input_index = i - j + k_radius;

            if (input_index >= 0 && input_index < n_arr) {
                if (!mask[input_index]) continue; // skip if mask is false
                out[i] += arr[input_index] * kernel[j];
            }
            else {
                out[i] += pad_val * kernel[j];
            }
        }
    }
}

void convolution_1d(
    int grid_size,
    int block_size,
    float* out,
    float* arr,
    int n_arr,
    float* kernel,
    int n_kernel,
    bool* mask,
    float pad_val) {
    convolution_1d_kernel << <grid_size, block_size, block_size + n_kernel >> > (
        out, arr, n_arr, kernel, n_kernel, mask, pad_val
        );
}

void convolution_1d_cpu(float* out, float* arr, int n_arr, float* kernel, int n_kernel, bool* mask, float pad_val) {

    for (int i = 0; i < n_arr; i++) {
        if (!mask[i]) continue; // skip if mask is false

        // loop over kernel
        for (int j = 0; j < n_kernel; ++j) {

            // index the array with implicit reversed kernel
            int input_index = i - j + n_kernel / 2;
            if (input_index >= 0 && input_index < n_arr) {
                if (!mask[input_index]) continue; // skip if mask is false
                out[i] += arr[input_index] * kernel[j];
            }
            else {
                out[i] += pad_val * kernel[j];
            }
        }
    }
}
